#include "hip/hip_runtime.h"
#include "BigInteger.h"

#include <iostream>
#include <string>

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op == "+") {
		return ADD;
	} else if (op == "-") {
		return SUBSTRACT;
	} else if (op == "*") {
		return MULTIPLY;
	} else if (op == "/") {
		return DIVIDE;
	} else if (op == "!") {
		return FACTORIAL;
	} else if (op == "pgcd") {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number(0), size(1) {}

/*BigInteger::BigInteger(const char* number, int size) : size(size) {

}*/


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete number;
	number = new char[size];
	for (int i = 0; i < size; i++) {
		number[i] = nuNumber[i];
	}
}

void BigInteger::zero() {
	for (int i = 0; i < size; i++) {
		number[i] -= '0';
	}
}

void bump(char* number, int size) {
	for (int i = 0; i < size; i++) {
		number[i] += '0';
	}
}




//init 
// size = size_biggest + diff +1 (addition case)
__device__ void init(int size, char* toFill) {
	for (int i = 0; i < size; i++) {
		toFill[i] = 0;
	}
}

//return size, update char*
__device__ int update(char* toUpdate, int value) {
	int dec = 0;
	for (int i = 0; i < value; i++) {
		if (toUpdate[i] == 0) {
			dec++;
		}
		else {
			break;
		}
	}

	int toReturn = value-dec;

	for (int i = 0; i < toReturn; i++) {
		toUpdate[i] = toUpdate[i+dec];
	}


	return toReturn;
}

/*
How to use this method :
- newB and size newB contains the result
- size_biggest is the biggest size
- diff is the difference between sizes
- first is the biggest char*
- second is the other
- calcul newB avant

*/

/*__global__*/ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int carry = 0;
	(*size_newB) = size_biggest + 1;
	init(*size_newB, newB);
	int index = *size_newB - 1;

/*
	// know 
	if (size_first > size_second) {
		size_biggest = size_second;
		diff = size_first - size_second;
	}
	else {
		size_biggest = size_first;
		diff = size_second - size_first;
	}

*/

	for (int i = size_biggest - 1; i >= 0; i--) {
		tmp = second[i] + first[i] + carry;
cout << "__ " << (int) first[i] << " + " << (int) second[i] << " + " << carry << " = " << tmp << endl;
		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
cout << index << "___ " << (int) newB[index] << endl;
		index--;
	}

	/*for (int i = diff - 1; i >= 0; i--) {
		tmp = first[i] + carry;
cout << "~~ " << (int) first[i] << " + " << carry << " = " << tmp << endl;
		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
cout << index << "~~~ " << (int) newB[index] << endl;
		index--;
	}*/
	if (carry != 0) {
		newB[index] = carry;
cout << index << "### " << (int) newB[index] << endl;
	}

cout << "cheking final result" << endl;
for (int i = 0; i < *size_newB; i++) {
	cout << (int) newB[i];
}
cout << endl;

}

int main(int argc, char** argv) {

	BigInteger left, right;
	OperationType opType;
	if (argc >= 2) {
		opType = identifyOperationType(argv[1]);
		left.setNumber(argv[2], string(argv[2]).size());
		left.zero();
		switch (opType) {
		case ADD:
		case SUBSTRACT:
		case MULTIPLY:
		case DIVIDE:
			right.setNumber(argv[3], string(argv[3]).size());
			right.zero();
			break;
		case ERROR:
			cout << "Unrecognised operation type: " << argv[1] << endl;
		}
	} else {
		cout << "Insufficient number of arguments" << endl;
	}

	BigInteger* g_left, *g_right;
	hipMalloc( (void**) &g_left, sizeof(BigInteger));
	hipMalloc( (void**) &g_right, sizeof(BigInteger));

	hipMemcpy(g_left, &left, sizeof(BigInteger), hipMemcpyHostToDevice);
	hipMemcpy(g_right, &right, sizeof(BigInteger), hipMemcpyHostToDevice);

	dim3 block(1), grid(1);
	///
	/// Testing block
	///
	char* nu = new char[2], * g_nu;
	char* first = new char[2];
	first[0] = 9; first[1] = 2;
	char* second = new char[2];
	second[0] = 1; second[1] = 9;
	int nuSize = 2;
	/*
	char* g_first, *g_second;
	hipMalloc( (void**) &g_first, sizeof(char) * 2 );
	hipMalloc( (void**) &g_second, sizeof(char) * 2 );
	hipMalloc( (void**) &g_nu, sizeof(char) * 2 );
	hipMemcpy(g_first, first, sizeof(char) * 2, hipMemcpyHostToDevice);
	hipMemcpy(g_second, second, sizeof(char) * 2, hipMemcpyHostToDevice);
	kernel_add<<<grid, block>>>(g_nu, g_first, g_second, 2, 2, &nuSize);
	hipMemcpy(nu, g_nu, sizeof(char) * 2, hipMemcpyDeviceToHost);
	*/
kernel_add(nu, first, second, 2, 2, &nuSize);
	cout << (int) first[0] << (int) first[1] << " + " << (int) second[0] << (int) second[1] << " = " << (int) nu[0] << (int) nu[1] << (int) nu[2]<< endl;
	///
	/// End of testing block
	///
	switch (opType) {
	case ADD:

		break;
	case SUBSTRACT:

		break;
	case MULTIPLY:

		break;
	case DIVIDE:

		break;
	case FACTORIAL:

		break;

	case GCD:

		break;
	default:
		cout << "Reaching default case." << endl;
		break;
	}

}

///
/// Reminder: Don't put functions or methods below the main
///

#include "BigInteger.h"

#include <iostream>
#include <string>

#include "utility.h"

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op[0] == '+') {
		return ADD;
	} else if (op[0] == '-') {
		return SUBSTRACT;
	} else if (op[0] == '*') {
		return MULTIPLY;
	} else if (op[0] == '/') {
		return DIVIDE;
	} else if (op[0] == '!') {
		return FACTORIAL;
	} else if (op == "pgcd") {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number(0), size(1) {}


BigInteger::BigInteger(int size) : size(size) {
	number = new char[size];
	init(size, number);
}


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete number;
	if (nuNumber[0] != '-' && nuNumber[0] != '+')
		size++;

	number = new char[size];
	if (nuNumber[0] == '-')
		number[0] = '-';
	else
		number[0] = '+';

	for (int i = 1; i < size; i++) {
		if (nuNumber[0] != '-' && nuNumber[0] != '+')
			number[i] = nuNumber[i - 1];
		else
			number[i] = nuNumber[i];
	}
}

void BigInteger::zero() {
	for (int i = 0; i < size; i++) {
		if (number[i] == '-' || number[i] == '+') continue;
		number[i] -= '0';
	}
}

void BigInteger::print() {
	cout << number[0];
	for (int i = 1; i < size; i++) {
		cout << (int) number[i] << "`";
	}
	cout << endl;
}

/**
 * Allocate and copy this BigInteger's number to device.
 */
char* BigInteger::copyNumberToDevice() const {
	char* d_number;
	hipMalloc( (void**) &d_number, sizeof(char) * size);
	hipMemcpy(d_number, number, sizeof(char) * size, hipMemcpyHostToDevice);
	return d_number;
}

void BigInteger::copyNumberFromDevice(char* d_number) {
	hipMemcpy(number, d_number, sizeof(char) * size, hipMemcpyDeviceToHost);
}

///
/// Carry the things
///

void BigInteger::applyAddCarry() {
	for (int i = size - 1; i >= 0; i--) {
		if (number[i] > 9) {
			number[i] -= 10;
			number[i - 1]++;
		}
	}
}

void BigInteger::applySubCarry() {
	for (int i = size - 1; i >= 0; i--) {
		if (number[i] < 0) {
			number[i] += 10;
			number[i - 1]--;
		}
	}
}

BigInteger BigInteger::applyMulCarry(int size_result, int size_first, int size_second) {
	int inc = 1;
	int i, n;
	char * tmp_result;
	BigInteger result(size_result);
	
	tmp_result = new char(size_result);

	for(i=0; i<size_result; i++)
		tmp_result[i] = 0;
	
	
	// i pour result, et n pour number
	for (i = size_result - 1, n = size_first*size_second-1; n > size_first; i--, n--) {
		if (n % size_first != 0) {
			tmp_result[i] += number[n];
			while (tmp_result[i]>9){
				tmp_result[i] -= 10;
				tmp_result[i-1]++;
			}
		}
		if(n % size_first == 0){
			i = size_result - inc;
			inc ++;
		}
	}
	// Signe du résultat
	tmp_result[0] = number[0];
	
	result.setNumber(tmp_result, size_result);
	return result;
}

///
/// Operations
///

BigInteger BigInteger::add(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();

	int size_b = size;
	if (other.size > size)
		size_b = other.size;
	
	BigInteger result(size_b + 1);
	char* d_newB = result.copyNumberToDevice();
	
	dim3 grid(1), block(size_b + 1);
	
	if (other.size > size) {
		kernel_add<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
	} else {
		kernel_add<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
	}
	hipFree(d_number);
	hipFree(d_other_number);

	result.copyNumberFromDevice(d_newB);
	result.applyAddCarry();
	return result;
}

BigInteger BigInteger::substract(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();

	int size_b = size;
	if (other.size > size)
		size_b = other.size;
	
	BigInteger result(size_b + 1);
	char* d_newB = result.copyNumberToDevice();
	
	dim3 grid(1), block(size_b + 1);

	kernel_sub<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);

	result.copyNumberFromDevice(d_newB);
	result.applySubCarry();
	return result;
}

BigInteger BigInteger::multiply(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();
	int size_newBbegin = size * other.size;
	
	BigInteger resultbegin(size_newBbegin);
	
	char* d_newB = resultbegin.copyNumberToDevice();
	
	dim3 grid(1), block(size, other.size);

	kernel_mul<<<grid, block>>>(d_newB, d_number, d_other_number, size, other.size, &size_newBbegin);

	resultbegin.copyNumberFromDevice(d_newB);

	hipFree(d_number);
	hipFree(d_other_number);
	hipFree(d_newB);

	int size_newB = size + other.size -1;
	BigInteger result(size_newB);
	result = resultbegin.applyMulCarry(size_newB, size, other.size);
	return result;
}

BigInteger BigInteger::divide(const BigInteger& other) {

}

BigInteger BigInteger::factorial(const BigInteger& other) {

}

BigInteger BigInteger::greatestCommonDivisor(const BigInteger& other) {

}




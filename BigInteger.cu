#include "hip/hip_runtime.h"
#include "BigInteger.h"

#include <iostream>
#include <string>

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op == "+") {
		return ADD;
	} else if (op == "-") {
		return SUBSTRACT;
	} else if (op == "*") {
		return MULTIPLY;
	} else if (op == "/") {
		return DIVIDE;
	} else if (op == "!") {
		return FACTORIAL;
	} else if (op == "pgcd") {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number(0), size(1) {}

/*BigInteger::BigInteger(const char* number, int size) : size(size) {

}*/


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete number;
	number = new char[size];
	for (int i = 0; i < size; i++) {
		number[i] = nuNumber[i];
	}
}

void BigInteger::zero() {
	for (int i = 0; i < size; i++) {
		number[i] -= '0';
	}
}

void bump(char* number, int size) {
	for (int i = 0; i < size; i++) {
		number[i] += '0';
	}
}




//init 
// size = size_biggest + diff +1 (addition case)
__device__ void init(int size, char* toFill) {
	for (int i = 0; i < size; i++) {
		toFill[i] = 0;
	}
}

//return size, update char*
__device__ int update(char* toUpdate, int value) {
	int dec = 0;
	for (int i = 0; i < value; i++) {
		if (toUpdate[i] == 0) {
			dec++;
		}
		else {
			break;
		}
	}

	int toReturn = value-dec;

	for (int i = 0; i < toReturn; i++) {
		toUpdate[i] = toUpdate[i+dec];
	}


	return toReturn;
}

/*
How to use this method :
- newB and size newB contains the result
- size_biggest is the biggest size
- diff is the difference between sizes
- first is the biggest char*
- second is the other
- calcul newB avant

*/

/*__global__*/ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int carry = 0;
	(*size_newB) = size_biggest + 1;
	init(*size_newB, newB);
	int index = *size_newB - 1;

/*
	// know 
	if (size_first > size_second) {
		size_biggest = size_second;
		diff = size_first - size_second;
	}
	else {
		size_biggest = size_first;
		diff = size_second - size_first;
	}

*/

	for (int i = size_biggest - 1; i >= 0; i--) {
		if (i - diff >= 0) {
			tmp = second[i - diff] + first[i] + carry;
			cout << "__ " << (int) first[i] << " + " << (int) second[i - diff] << " + " << carry << " = " << tmp << endl;
		} else {
			tmp = first[i] + carry;
			cout << "__ " << (int) first[i] << " + " << carry << " = " << tmp << endl;
		}

		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
		cout << index << "___ " << (int) newB[index] << endl;
		index--;
	}

	/*for (int i = diff - 1; i >= 0; i--) {
		tmp = first[i] + carry;
cout << "~~ " << (int) first[i] << " + " << carry << " = " << tmp << endl;
		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
cout << index << "~~~ " << (int) newB[index] << endl;
		index--;
	}*/
	if (carry != 0) {
		newB[index] = carry;
		cout << index << "### " << (int) newB[index] << endl;
	}

	cout << "cheking final result" << endl;
	for (int i = 0; i < *size_newB; i++) {
		cout << (int) newB[i];
	}
	cout << endl;

}

int main(int argc, char** argv) {

	BigInteger left, right;
	OperationType opType;
	if (argc >= 2) {
		opType = identifyOperationType(argv[1]);
		left.setNumber(argv[2], string(argv[2]).size());
		left.zero();
		switch (opType) {
		case ADD:
		case SUBSTRACT:
		case MULTIPLY:
		case DIVIDE:
			right.setNumber(argv[3], string(argv[3]).size());
			right.zero();
			break;
		case ERROR:
			cout << "Unrecognised operation type: " << argv[1] << endl;
		}
	} else {
		cout << "Insufficient number of arguments" << endl;
	}

	BigInteger* g_left, *g_right;
	hipMalloc( (void**) &g_left, sizeof(BigInteger));
	hipMalloc( (void**) &g_right, sizeof(BigInteger));

	hipMemcpy(g_left, &left, sizeof(BigInteger), hipMemcpyHostToDevice);
	hipMemcpy(g_right, &right, sizeof(BigInteger), hipMemcpyHostToDevice);

	dim3 block(1), grid(1);
	///
	/// Testing block
	///
	#define SIZE_FIRST 2
	#define SIZE_SECOND 1
	#define NU_SIZE 3
	char* nu = new char[NU_SIZE], * g_nu;
	char* first = new char[SIZE_FIRST];
	first[0] = 9; first[1] = 5;
	char* second = new char[SIZE_SECOND];
	second[0] = 7;
	int nuSize = NU_SIZE;
	/*
	char* g_first, *g_second;
	hipMalloc( (void**) &g_first, sizeof(char) * 2 );
	hipMalloc( (void**) &g_second, sizeof(char) * 2 );
	hipMalloc( (void**) &g_nu, sizeof(char) * 2 );
	hipMemcpy(g_first, first, sizeof(char) * 2, hipMemcpyHostToDevice);
	hipMemcpy(g_second, second, sizeof(char) * 2, hipMemcpyHostToDevice);
	kernel_add<<<grid, block>>>(g_nu, g_first, g_second, 2, 2, &nuSize);
	hipMemcpy(nu, g_nu, sizeof(char) * 2, hipMemcpyDeviceToHost);
	*/
	kernel_add(nu, first, second, SIZE_FIRST, SIZE_FIRST - SIZE_SECOND, &nuSize);
	for (int i = 0; i < SIZE_FIRST; i++) {
		cout << (int) first[i];
	}
	cout << " + ";
	for (int i = 0; i < SIZE_SECOND; i++) {
		cout << (int) second[i];
	}
	cout << " = ";
	for (int i = 0; i < nuSize; i++) {
		cout << (int) nu[i];
	}
	cout << endl;
	///
	/// End of testing block
	///
	switch (opType) {
	case ADD:

		break;
	case SUBSTRACT:

		break;
	case MULTIPLY:

		break;
	case DIVIDE:

		break;
	case FACTORIAL:

		break;

	case GCD:

		break;
	default:
		cout << "Reaching default case." << endl;
		break;
	}

}

///
/// Reminder: Don't put functions or methods below the main
///

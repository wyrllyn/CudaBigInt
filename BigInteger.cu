#include "hip/hip_runtime.h"
#include "BigInteger.h"

#include <iostream>
#include <string>

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op == "+") {
		return ADD;
	} else if (op == "-") {
		return SUBSTRACT;
	} else if (op == "*") {
		return MULTIPLY;
	} else if (op == "/") {
		return DIVIDE;
	} else if (op == "!") {
		return FACTORIAL;
	} else if (op == "pgcd") {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number(0), size(1) {}

/*BigInteger::BigInteger(const char* number, int size) : size(size) {

}*/


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete number;
	number = new char[size];
	for (int i = 0; i < size; i++) {
		number[i] = nuNumber[i];
	}
}

void BigInteger::zero() {
	for (int i = 0; i < size; i++) {
		number[i] -= '0';
	}
}

bool BigInteger::isNeg(char* bi) {
	if (bi[0] == '-') {
		return true;
	}
	else
		return false;
}

void bump(char* number, int size) {
	for (int i = 0; i < size; i++) {
		number[i] += '0';
	}
}

__device__ void add_minus(char* bi, int size) {
	for (int i = size; i > 0 ; i--) {
		bi[i] = bi[i - 1];
	}
	bi[0] = '-';
}
__device__ void remove_minus(char* bi, int size) {
	for (int i = 0; i < size ; i++) {
		bi[i] = bi[i+1];
	}
}




//init 
// size = size_biggest + diff +1 (addition case)
__device__ void init(int size, char* toFill) {
	for (int i = 0; i < size; i++) {
		toFill[i] = 0;
	}
}

//return size, update char*
__device__ int update(char* toUpdate, int value) {
	int dec = 0;
	for (int i = 0; i < value; i++) {
		if (toUpdate[i] == 0) {
			dec++;
		}
		else {
			break;
		}
	}

	int toReturn = value-dec;

	for (int i = 0; i < toReturn; i++) {
		toUpdate[i] = toUpdate[i+dec];
	}

	if (toReturn == 0) {
		toReturn++;
		toUpdate[0] = 0;
	}


	return toReturn;
}

/**
 * Returns true if first is bigger or equal to second.
 * Note: assumes that both numbers have the same size.
 */
int isFirstBiggerThanSecond(const char* first, const char* second, int size) {
	for (int i = 0; i < size; i++) {
		if (first[i] > second[i]) return 1;
		else if (first[i] < second[i]) return 0;
		else continue;
	}
	return 1;
}

/*
How to use this method :
- newB and size newB contains the result
- size_biggest is the biggest size
- diff is the difference between sizes
- first is the biggest char*
- second is the other
- calcul newB avant

*/

/*__global__*/ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int carry = 0;
	(*size_newB) = size_biggest + 1;
	init(*size_newB, newB);
	int index = *size_newB - 1;

/*
	// know 
	if (size_first > size_second) {
		size_biggest = size_second;
		diff = size_first - size_second;
	}
	else {
		size_biggest = size_first;
		diff = size_second - size_first;
	}

*/

	for (int i = size_biggest - 1; i >= 0; i--) {
		if (i - diff >= 0) {
			tmp = second[i - diff] + first[i] + carry;
			cout << "__ " << (int) first[i] << " + " << (int) second[i - diff] << " + " << carry << " = " << tmp << endl;
		} else {
			tmp = first[i] + carry;
			cout << "__ " << (int) first[i] << " + " << carry << " = " << tmp << endl;
		}

		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
		cout << index << "___ " << (int) newB[index] << endl;
		index--;
	}

	/*for (int i = diff - 1; i >= 0; i--) {
		tmp = first[i] + carry;
cout << "~~ " << (int) first[i] << " + " << carry << " = " << tmp << endl;
		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
cout << index << "~~~ " << (int) newB[index] << endl;
		index--;
	}*/
	if (carry != 0) {
		newB[index] = carry;
		cout << index << "### " << (int) newB[index] << endl;
	}

	cout << "cheking final result" << endl;
	for (int i = 0; i < *size_newB; i++) {
		cout << (int) newB[i];
	}
	cout << endl;

}

/**
 * first is divided by second.
 * Note: since we are working with integers, the quotient can't be bigger than the dividend.
 * Also, if the divisor is bigger than the dividend, then the result is zero.
 */
void kernel_div(char* newB, const char* first, const char* second, int size_first, int size_second, int * size_newB) {
	if (size_first > size_second
		|| (size_first == size_second && isFirstBiggerThanSecond(first, second, size_first))
	) {
		*size_newB = size_first;
	} else {
		*size_newB = 1;
		init(*size_newB, newB);
		return;
	}


	char* temp = NULL;
	init(size_second + 1, temp);
	int t = 0; // temp's index
	int n = 0; // newB's index
	for (int i = size_first - 1; i >= 0; i -= t) {
		t = 0;
		for (int j = i - size_second; j <= i; j++) {
			if (j >= 0) {
				temp[t] = first[j];
				t++;
			}
		}
		// verify that we are not attempting to divide something too small
		if (isFirstBiggerThanSecond(second, temp, size_second)) {
			t = 0;
			for (int j = i - size_second - 1; j <= i; j++) {
				if (j < 0) {
					// nothing left to divide, exit function
					return;
				} else {
					temp[t] = first[j];
					t++;
				}
			}
		}
		// now that we have our thing, let's get to the division itself
		char res = 0;
		char* sub_res = NULL;
		int size_res = 0;
		init(size_second, sub_res);
		do {		
			//kernel_sub(sub_res, temp, second, size_second, size_second, &size_res);
			res++;
		} while (0); //sub_res > 0
		// current division done, save result & move on to the next
		newB[n] = res;
		n++;
	}
	// all divisions done, we need to realign our result;
	int diff = size_second - n;
	for (int i = size_second - 1; i > n; i++) {
		newB[i] = newB[i - diff];
	}
}

/*__global__*/ void kernel_sub(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {

	int tmp = 0;
	int carry = 0;
	(*size_newB) = size_biggest;
	init(*size_newB, newB);
	int index = *size_newB - 1;

	for (int i = size_biggest - 1; i >= 0; i--) {
		if (i - diff >= 0) {
			tmp = first[i] - second[i-diff] - carry;
			//cout << "__ " << (int) first[i] << " - " << (int) second[i - diff] << " - " << carry << " = " << tmp << endl;
		} else {
			tmp = first[i] - carry;
			//cout << "__ " << (int) first[i] << " - " << carry << " = " << tmp << endl;
		}

		if (tmp < 0) {
			// warning 10 - tmp ?
			carry = 1;
			tmp += 10 ;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
		cout << "index : " << index << "___ " << (int) newB[index] << endl;
		index--;
	}

	*size_newB = update(newB, *size_newB);
	

	cout << "cheking final result" << endl;
	for (int i = 0; i < *size_newB; i++) {
		cout << (int) newB[i];
	}
	cout << endl;
}

// first is the bigInt with de biggest size
/*__global__*/ void kernel_mul(char* newB,  char* first, char* second, int size_first, int size_second, int * size_newB) {
	(*size_newB) = size_first + size_second;
	init(*size_newB, newB);
	int index = 0;
	int tmp = 0;
	int tmp_second = 0;
	int carry = 0;
	int carry_second = 0;

	for (int i = size_second - 1; i >= 0; i--) {
		index = (*size_newB) - size_second + i ;
		for (int j = size_first - 1; j >= 0 ; j--) {
			//cout << "i = " << i << " j= " << j << " index = " << index << endl;
			tmp = first[j] * second[i] + carry;
			//cout << "1 tmp = " << tmp << endl;
			carry = 0;
			while (tmp >= 10) {
				tmp -= 10;
				carry++;
			}
		//	cout << "2 tmp = " << tmp << endl;
		//	cout << "carry = " << carry << endl;

			tmp_second = newB[index] + tmp + carry_second;
			if (tmp_second >= 10) {
		//		cout << " test second " << endl;
				tmp_second = tmp_second % 10;
				carry_second = 1;
			}
			newB[index] = tmp_second;
			index --;
			if (carry > 0 && j == 0) {
		//		cout << "test" << endl;
				newB[index] = carry;
			}
		}


		// add values : how ??
	}
	
}

int main(int argc, char** argv) {

	BigInteger left, right;
	OperationType opType;
	if (argc >= 2) {
		opType = identifyOperationType(argv[1]);
		left.setNumber(argv[2], string(argv[2]).size());
		left.zero();
		switch (opType) {
		case ADD:
		case SUBSTRACT:
		case MULTIPLY:
		case DIVIDE:
			right.setNumber(argv[3], string(argv[3]).size());
			right.zero();
			break;
		case ERROR:
			cout << "Unrecognised operation type: " << argv[1] << endl;
		}
	} else {
		cout << "Insufficient number of arguments" << endl;
	}

	BigInteger* g_left, *g_right;
	hipMalloc( (void**) &g_left, sizeof(BigInteger));
	hipMalloc( (void**) &g_right, sizeof(BigInteger));

	hipMemcpy(g_left, &left, sizeof(BigInteger), hipMemcpyHostToDevice);
	hipMemcpy(g_right, &right, sizeof(BigInteger), hipMemcpyHostToDevice);

	dim3 block(1), grid(1);
	///
	/// Testing block
	///
	#define SIZE_FIRST 2
	#define SIZE_SECOND 2
	#define NU_SIZE 4
	char* nu = new char[NU_SIZE], * g_nu;
	char* first = new char[SIZE_FIRST];
	first[0] = 3; first[1] = 5;
	char* second = new char[SIZE_SECOND];
	second[0] = 2; second[1] = 0; // second[2] = 2;
	int nuSize = NU_SIZE;
	/*
	char* g_first, *g_second;
	hipMalloc( (void**) &g_first, sizeof(char) * 2 );
	hipMalloc( (void**) &g_second, sizeof(char) * 2 );
	hipMalloc( (void**) &g_nu, sizeof(char) * 2 );
	hipMemcpy(g_first, first, sizeof(char) * 2, hipMemcpyHostToDevice);
	hipMemcpy(g_second, second, sizeof(char) * 2, hipMemcpyHostToDevice);
	kernel_add<<<grid, block>>>(g_nu, g_first, g_second, 2, 2, &nuSize);
	hipMemcpy(nu, g_nu, sizeof(char) * 2, hipMemcpyDeviceToHost);
	*/
	//kernel_add(nu, first, second, SIZE_FIRST, SIZE_FIRST - SIZE_SECOND, &nuSize);
	//kernel_div(nu, first, second, SIZE_FIRST, SIZE_FIRST - SIZE_SECOND, &nuSize);
	kernel_mul(nu, first, second, SIZE_FIRST, /*SIZE_FIRST -*/ SIZE_SECOND, &nuSize);
	for (int i = 0; i < SIZE_FIRST; i++) {
		cout << (int) first[i];
	}
	cout << " * ";
	for (int i = 0; i < SIZE_SECOND; i++) {
		cout << (int) second[i];
	}
	cout << " = ";
	for (int i = 0; i < nuSize; i++) {
		cout << (int) nu[i];
	}
	cout << endl;
	///
	/// End of testing block
	///
	switch (opType) {
	case ADD:

		break;
	case SUBSTRACT:

		break;
	case MULTIPLY:

		break;
	case DIVIDE:

		break;
	case FACTORIAL:

		break;

	case GCD:

		break;
	default:
		cout << "Reaching default case." << endl;
		break;
	}

}

///
/// Reminder: Don't put functions or methods below the main
///

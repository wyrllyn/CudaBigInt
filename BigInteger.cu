#include "BigInteger.h"

#include <iostream>
#include <string>

#include "utility.h"

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op[0] == '+') {
		return ADD;
	} else if (op[0] == '-') {
		return SUBSTRACT;
	} else if (op[0] == '*') {
		return MULTIPLY;
	} else if (op[0] == '/') {
		return DIVIDE;
	} else if (op[0] == '!') {
		return FACTORIAL;
	} else if (op == "pgcd") {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number(0), size(1) {}


BigInteger::BigInteger(int size) : size(size) {
	number = new char[size];
	init(size, number);
}


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete number;
	if (nuNumber[0] != '-' && nuNumber[0] != '+')
		size++;

	number = new char[size];
	if (nuNumber[0] == '-')
		number[0] = '-';
	else
		number[0] = '+';

	for (int i = 1; i < size; i++) {
		if (nuNumber[0] != '-' && nuNumber[0] != '+')
			number[i] = nuNumber[i - 1];
		else
			number[i] = nuNumber[i];
	}
}

void BigInteger::zero() {
	for (int i = 0; i < size; i++) {
		if (number[i] == '-' || number[i] == '+') continue;
		number[i] -= '0';
	}
}

void BigInteger::print() {
	cout << number[0];
	for (int i = 1; i < size; i++) {
		cout << (int) number[i];
	}
	cout << endl;
}

/**
 * Allocate and copy this BigInteger's number to device.
 */
char* BigInteger::copyNumberToDevice() const {
	char* d_number;
	hipMalloc( (void**) &d_number, sizeof(char) * size);
	hipMemcpy(d_number, number, sizeof(char) * size, hipMemcpyHostToDevice);
	return d_number;
}

/**
 * TODO
 */
void BigInteger::copyNumberFromDevice(char* d_number) {
	hipMemcpy(number, d_number, sizeof(char) * size, hipMemcpyDeviceToHost);
}

BigInteger BigInteger::add(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();

	int size_b = size;
	if (other.size > size)
		size_b = other.size;
	
	BigInteger result(size_b + 1);
	char* d_newB = result.copyNumberToDevice();
	

cout << "size_b = " << size_b << endl;
	dim3 grid(1), block(size_b + 1);
	
	if (other.size > size) {
		kernel_add<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
	} else {
		kernel_add<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
	}

	result.copyNumberFromDevice(d_newB);
	return result;
}

BigInteger BigInteger::substract(const BigInteger& other) {

}

BigInteger BigInteger::multiply(const BigInteger& other) {

}

BigInteger BigInteger::divide(const BigInteger& other) {

}

BigInteger BigInteger::factorial(const BigInteger& other) {

}

BigInteger BigInteger::greatestCommonDivisor(const BigInteger& other) {

}




#include "BigInteger.h"

#include <iostream>
#include <string>

#include "utility.h"

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op[0] == '+') {
		return ADD;
	} else if (op[0] == '-') {
		return SUBSTRACT;
	} else if (op[0] == '*') {
		return MULTIPLY;
	} else if (op[0] == '/') {
		return DIVIDE;
	} else if (op[0] == '!') {
		return FACTORIAL;
	} else if (op == "pgcd") {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number(0), size(1) {}


BigInteger::~BigInteger() {
	//delete [] number;
}


BigInteger::BigInteger(int size) : size(size) {
	number = new char[size];
	init(size, number);
	number[0] = '+';
}


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete [] number;
	if (nuNumber[0] != '-' && nuNumber[0] != '+')
		size++;

	number = new char[size];
	if (nuNumber[0] == '-')
		number[0] = '-';
	else
		number[0] = '+';

	for (int i = 1; i < size; i++) {
		if (nuNumber[0] != '-' && nuNumber[0] != '+')
			number[i] = nuNumber[i - 1];
		else
			number[i] = nuNumber[i];
	}
}

void BigInteger::zero() {
	for (int i = 0; i < size; i++) {
		if (number[i] == '-' || number[i] == '+') continue;
		number[i] -= '0';
	}
}

void BigInteger::print() const {
	cout << number[0];
	cout.flush();
	for (int i = 1; i < size; i++) {
		cout << "`"  << (int) number[i] ;
	}
	cout << endl;
}

void BigInteger::reset() {
	init(size, number);
	number[0] = '+';
}

void BigInteger::resize() {
	int dec = 0;
	for (int i = 1; i < size; i++) {
		if (number[i] == 0) {
			dec++;
		}
		else {
			break;
		}
	}
	if (size - dec <= 1) {
		number[0] = '+';
		number[1] = 0;
		size = 2;
	}
	else if (dec != 0) {
		for (int i = 1; i < size - dec; i++) {
			number[i] = number[i + dec];
		}
		size -= dec;
	}
}

/**
 * Allocate and copy this BigInteger's number to device.
 */
char* BigInteger::copyNumberToDevice() const {
	char* d_number;
	hipMalloc( (void**) &d_number, sizeof(char) * size);
	hipMemcpy(d_number, number, sizeof(char) * size, hipMemcpyHostToDevice);
	return d_number;
}


void BigInteger::copyNumberFromDevice(char* d_number) {
	hipMemcpy(number, d_number, sizeof(char) * size, hipMemcpyDeviceToHost);
}

///
/// Carry the things
///

void BigInteger::applyAddCarry() {
	for (int i = size - 1; i >= 1; i--) {
		if (number[i] > 9) {
			number[i] -= 10;
			number[i - 1]++;
		}
	}
}

void BigInteger::applySubCarry() {
	for (int i = size - 1; i >= 0; i--) {
		if (number[i] < 0) {
			number[i] += 10;
			number[i - 1]--;
		}
	}
}

BigInteger BigInteger::applyMulCarry(int size_result, int size_first, int size_second) {
	int inc = 1;
	int i, n;
	char * tmp_result;
	BigInteger result(size_result);
	
	tmp_result = new char[size_result];

	for(i=1; i<size_result; i++)
		tmp_result[i] = 0;
	
	
	// i pour result, et n pour number
	for (i = size_result - 1, n = size_first*size_second-1; n > size_first; i--, n--) {
		if (n % size_first != 0) {
			tmp_result[i] += number[n];
			while (tmp_result[i]>9){
				tmp_result[i] -= 10;
				tmp_result[i-1]++;
			}
		}
		if(n % size_first == 0){
			i = size_result - inc;
			inc ++;
		}
	}
	// Signe du résultat
	tmp_result[0] = number[0];
	
	result.setNumber(tmp_result, size_result);

	delete [] tmp_result;
	return result;
}

///
/// Operations
///

BigInteger BigInteger::add(const BigInteger& other) {

	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();

	int size_b = size;
	if (other.size > size)
		size_b = other.size;
	
	BigInteger result(size_b + 1);
	char* d_newB = result.copyNumberToDevice();
	
	dim3 grid(1), block(size_b + 1);

	if (number[0] == other.number[0]) {	
		if (other.size > size) {
			kernel_add<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
		} else {
			kernel_add<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
		}
		result.copyNumberFromDevice(d_newB);
		result.applyAddCarry();
		if (number[0] == '-') {
			result.number[0] = '-';
		}
		else {
			result.number[0] = '+';
		}
	}
	else {
		// check where minus is
		if(other.number[0] == '-') {
			// check size of the number with minus
			if (isFirstBiggerThanSecond_2(number, other.number, size, other.size)) {
				kernel_sub<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
				result.copyNumberFromDevice(d_newB);
				result.applySubCarry();
			}
			else {
				kernel_sub<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
				result.copyNumberFromDevice(d_newB);
				result.applySubCarry();
				result.number[0] = '-';
			}

		}
		else {
			// check size of the number with minus
			if (isFirstBiggerThanSecond_2(number, other.number, size, other.size)) {
				kernel_sub<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
				result.copyNumberFromDevice(d_newB);
				result.applySubCarry();
				result.number[0] = '-';
			}
			else {
				kernel_sub<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
				result.copyNumberFromDevice(d_newB);
				result.applySubCarry();
			}
		}
	}

	result.resize();
	return result;
}

BigInteger BigInteger::substract(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();

	int size_b = size;
	if (other.size > size)
		size_b = other.size;
	
	BigInteger result(size_b + 1);
	char* d_newB = result.copyNumberToDevice();
	
	dim3 grid(1), block(size_b + 1);

	if (number[0] != other.number[0]) {
		if (isFirstBiggerThanSecond_2(number, other.number, size, other.size))
			kernel_add<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
		else
			kernel_add<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);

		result.copyNumberFromDevice(d_newB);
		result.applyAddCarry();
		if (number[0] == '-') {
			result.number[0] = '-';
		}
		else
				result.number[0] = '+';
	}
	else {
		if(isFirstBiggerThanSecond_2(number, other.number, size, other.size)) {
			kernel_sub<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
			result.copyNumberFromDevice(d_newB);
			result.applySubCarry();
			if (number[0] == '-') {
				result.number[0] = '-';
			}
			else
				result.number[0] = '+';
		}
		else {
			kernel_sub<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
			result.copyNumberFromDevice(d_newB);
			result.applySubCarry();
			if (number[0] == '+') {
				result.number[0] = '-';
			}
			else
				result.number[0] = '+';
		}
	}
	result.resize();
	return result;
}

BigInteger BigInteger::multiply(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();
	int size_newBbegin = size * other.size;
	
	BigInteger resultbegin(size_newBbegin);
	
	char* d_newB = resultbegin.copyNumberToDevice();
	
	dim3 grid(1), block(size, other.size);

	kernel_mul<<<grid, block>>>(d_newB, d_number, d_other_number, size, other.size, &size_newBbegin);

	resultbegin.copyNumberFromDevice(d_newB);

	hipFree(d_number);
	hipFree(d_other_number);
	hipFree(d_newB);

	int size_newB = size + other.size -1;
	BigInteger result(size_newB);
	result = resultbegin.applyMulCarry(size_newB, size, other.size);
	result.resize();
	return result;
}

BigInteger BigInteger::divide(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();
	int size_newB = size;
	if (other.size > size) {
		return BigInteger(0);
	}

	BigInteger result(size_newB);
	if(number[0]=='-' || other.number[0]=='-')
		result.number[0]='-';
	else
		result.number[0]='+';

	dim3 grid(1), block();

	//char* temp = new char[other.size + 1];
	//init(other.size + 1, temp);
	int t = 0; // temp's index
	int n = 0; // newB's index
	for (int i = size - 1; i > 0; i -= t) {
		t = 0;
		BigInteger temp(other.size + 1);
		cout << "loop#" << i << endl;
		for (int j = i - other.size - 1; j <= i; j++) {
			if (j > 0) {
				temp.number[1 + t] = number[j];
				t++;
				if (!isFirstBiggerThanSecond(other.number, temp.number, other.size)) {
					break;
				}
			}
		}
		cout<<"first phase done"<<endl;
		temp.print();
		// verify that we are not attempting to divide something too small
		bool stopDividing = false;
		if (isFirstBiggerThanSecond(other.number, temp.number, other.size)) {
			cout << "expanding our horizons"<<endl;
			for (int j = t + 1; j < other.size; j++) {
				temp.number[1 + t] = number[j];
				t++;
				if (!isFirstBiggerThanSecond(other.number, temp.number, other.size)) {
					break;
				}
			}
			// we went through the entire number & it's still not enough, that means we're done
			if (!isFirstBiggerThanSecond(other.number, temp.number, other.size)) {
				stopDividing = true; // not breaking right here 'cause we've got a print below
			}
		} else {
			// if everything is cool, shift right
			cout<<"shifting right "<<temp.size<<endl;
			for (int k = temp.size - 1; k > 1; k--) {
				cout<<"k="<<k<<endl;
				temp.number[k] = temp.number[k - 1];
			}
			temp.number[1] = 0;
		}
		temp.print();
		if (stopDividing) break;
		// now that we have our thing, let's get to the division itself
		char res = 0;
		//char* sub_res = new char[size_second];
		BigInteger sub_res(other.size);
		//init(size_second, sub_res);
		do {		
			//kernel_sub(sub_res, temp, second, size_second, size_second, &size_res);
			sub_res = temp.substract(other);
			if (sub_res.number[0] == '+') {
				res++;
			}
			temp = sub_res;
		} while (sub_res.number[0] == '+'); //sub_res > 0
		// current division done, save result & move on to the next
		n++;
		result.number[n] = res;
		cout <<"just checkin: i="<<i<<";t="<<t<<";n="<<n<<";res="<<(int)res<<endl;
	}
	// all divisions done, we need to realign our result;
	int diff = other.size - n + 1;
	cout<<"diff="<<diff<<endl;
	cout<<"size-1="<<(size - 1)<<endl;
	result.print();
	for (int i = size - 1; i > 0; i--) {
		if (i > n && (i - diff) > 0) {
			cout<<"##"<<i<<"="<<(int)result.number[i - diff]<<endl;
			result.number[i] = result.number[i - diff];
		} else {
			result.number[i] = 0;
		}
	}


	return result;
}

BigInteger BigInteger::factorial(const BigInteger& other) {
	return BigInteger(0);
}

BigInteger BigInteger::greatestCommonDivisor(const BigInteger& other) {
	return BigInteger(0);
}




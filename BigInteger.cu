#include "BigInteger.h"

#include <iostream>
#include <string>

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op == "+") {
		return ADD;
	} else if (op == "-") {
		return SUBSTRACT;
	} else if (op == "*") {
		return MULTIPLY;
	} else if (op == "/") {
		return DIVIDE;
	} else if (op == "!") {
		return FACTORIAL;
	} else if (op == "pgcd") {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number(0), size(1) {}

/*BigInteger::BigInteger(const char* number, int size) : size(size) {

}*/


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete number;
	number = new char[size];
	for (int i = 0; i < size; i++) {
		number[i] = nuNumber[i];
	}
}

void BigInteger::zero() {
	for (int i = 0; i < size; i++) {
		number[i] -= '0';
	}
}


int main(int argc, char** argv) {

	BigInteger left, right;
	OperationType opType;
	if (argc >= 2) {
		opType = identifyOperationType(argv[1]);
		left.setNumber(argv[2], string(argv[2]).size());
		left.zero();
		switch (opType) {
		case ADD:
		case SUBSTRACT:
		case MULTIPLY:
		case DIVIDE:
			right.setNumber(argv[3], string(argv[3]).size());
			right.zero();
			break;
		case ERROR:
			cout << "Unrecognised operation type: " << argv[1] << endl;
		}
	} else {
		cout << "Insufficient number of arguments" << endl;
	}

	BigInteger* g_left, *g_right;
	hipMalloc( (void**) &g_left, sizeof(BigInteger));
	hipMalloc( (void**) &g_right, sizeof(BigInteger));

	hipMemcpy(g_left, &left, sizeof(BigInteger), hipMemcpyHostToDevice);
	hipMemcpy(g_right, &right, sizeof(BigInteger), hipMemcpyHostToDevice);

	dim3 block, grid;
	switch (opType) {
	case ADD:

		break;
	case SUBSTRACT:

		break;
	case MULTIPLY:

		break;
	case DIVIDE:

		break;
	case FACTORIAL:

		break;

	case GCD:

		break;
	default:
		cout << "Reaching default case." << endl;
		break;
	}

}

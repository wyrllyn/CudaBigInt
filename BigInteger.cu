#include "hip/hip_runtime.h"
#include "BigInteger.h"

#include <iostream>
#include <string>

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op == "+") {
		return ADD;
	} else if (op == "-") {
		return SUBSTRACT;
	} else if (op == "*") {
		return MULTIPLY;
	} else if (op == "/") {
		return DIVIDE;
	} else if (op == "!") {
		return FACTORIAL;
	} else if (op == "pgcd") {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number("0"), size(1) {}

/*BigInteger::BigInteger(const char* number, int size) : size(size) {

}*/


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete number;
	number = new char[size];
	for (int i = 0; i < size; i++) {
		number[i] = nuNumber[i];
	}
}


int main(int argc, char** argv) {

	BigInteger left, right;
	OperationType opType;
	if (argc >= 2) {
		opType = identifyOperationType(argv[1]);
		left.setNumber(argv[2], string(argv[2]).size());
		switch (opType) {
		case ADD:
		case SUBSTRACT:
		case MULTIPLY:
		case DIVIDE:
			right.setNumber(argv[3], string(argv[3]).size());
			break;
		case ERROR:
			cout << "Unrecognised operation type: " << argv[1] << endl;
		}
	} else {
		cout << "Insufficient number of arguments" << endl;
	}

	BigInteger* g_left, *g_right;
	hipMalloc( (void**) &g_left, sizeof(BigInteger));
	hipMalloc( (void**) &g_right, sizeof(BigInteger));

	hipMemcpy(g_left, &left, sizeof(BigInteger), hipMemcpyHostToDevice);
	hipMemcpy(g_right, &right, sizeof(BigInteger), hipMemcpyHostToDevice);

	dim3 block, grid;
	switch (opType) {
	case ADD:

		break;
	case SUBSTRACT:

		break;
	case MULTIPLY:

		break;
	case DIVIDE:

		break;
	case FACTORIAL:

		break;

	case GCD:

		break;
	default:
		cout << "Reaching default case." << endl;
		break;
	}

}



//init 
// size = vali + hjvo +1 (addition case)
__device__ void init(int size, char* toFill) {
	for (int i = 0; i < size; i++) {
		toFill[i] = 0;
	}
}

//return size, update char*
__device__ int update(char* toUpdate, int value) {
	int dec = 0;
	for (int i = 0; i < value; i++) {
		if (toUpdate[i] == 0) {
			dec++;
		}
		else {
			break;
		}
	}

	int toReturn = value-dec;

	for (int i = 0; i < toReturn; i++) {
		toUpdate[i] = toUpdate[i+dec];
	}


	return toReturn;
}

/*
How to use this method :
- newB and size newB contains the result
- vali is the biggest size
- hjvo is the difference between sizes
- first is the biggest char*
- second is the other
- calcul newB avant

*/

__global__ void kernel_add(char* newB, char* first, char* second, int vali, int hjvo, int * size_newB) {
	int tmp = 0;
	int carry = 0;
	(*size_newB) = vali + hjvo +1;
	init(*size_newB, newB);
	int index = *size_newB;

/*
	// know 
	if (size_first > size_second) {
		vali = size_second;
		hjvo = size_first - size_second;
	}
	else {
		vali = size_first;
		hjvo = size_second - size_first;
	}

*/

	for (int i = vali - 1; i >= 0; i--) {
		tmp = second[i] + first[i] + carry;
		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
		index--;
	}

	for (int i = hjvo - 1; i >= 0; i--) {
		tmp = first[i] + carry;
		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
		index--;
	}
	if (carry != 0) {
		newB[index] = carry;
	}

}

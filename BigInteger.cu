#include "BigInteger.h"

#include <iostream>
#include <string>

#include "utility.h"

using namespace std;

OperationType identifyOperationType(const char* op) {
	if (op[0] == '+') {
		return ADD;
	} else if (op[0] == '-') {
		return SUBSTRACT;
	} else if (op[0] == '*') {
		return MULTIPLY;
	} else if (op[0] == '/') {
		return DIVIDE;
	} else if (op[0] == '!') {
		return FACTORIAL;
	} else if (op[0] == 'p') {
		return GCD;
	} else {
		return ERROR;
	}
}

BigInteger::BigInteger() : number(0), size(1) {}

BigInteger::~BigInteger() {
	//delete [] number;
}


BigInteger::BigInteger(int size) : size(size) {
	number = new char[size];
	init(size, number);
	number[0] = '+';

}


void BigInteger::setNumber(const char* nuNumber, int nuSize) {
	size = nuSize;
	delete [] number;
	if (nuNumber[0] != '-' && nuNumber[0] != '+')
		size++;

	number = new char[size];
	if (nuNumber[0] == '-')
		number[0] = '-';
	else
		number[0] = '+';

	for (int i = 1; i < size; i++) {
		if (nuNumber[0] != '-' && nuNumber[0] != '+')
			number[i] = nuNumber[i - 1];
		else
			number[i] = nuNumber[i];
	}
}

void BigInteger::zero() {
	for (int i = 0; i < size; i++) {
		if (number[i] == '-' || number[i] == '+') continue;
		number[i] -= '0';
	}
}


void BigInteger::print() const {
	cout << number[0];
	cout.flush();
	for (int i = 1; i < size; i++) {
		cout << "`"  << (int) number[i] ;
	}
	cout << endl;
}

void BigInteger::reset() {
	init(size, number);
	number[0] = '+';
}

void BigInteger::shiftRight(int offset) {
	for (int i = size - 1; i > offset; i--) {
		number[i] = number[i - offset];
	}
	for (int i = 1; i <= offset; i++) {
		number[i] = 0;
	}
}

///
/// Utility methods
///

void BigInteger::alignLeft(int* nuSize) {
	for (int i = 1; i < size; i++) {
		if (number[i] != 0) {
			//cout << "align: " << i << endl;
			//cout << "nuSize: " << (size - i) << endl;
			*nuSize = size - i;
			break;
		}
	}
	int i = 1;
	for (int j = size - *nuSize; j < size; j++) {
		//cout << "i=" << i << ";j=" << j << endl;
		//cout << "    =" << (int)number[j] << endl;
		number[i] = number[j];
		i++;
	}
	for (int i = *nuSize + 1; i < size; i++) {
		number[i] = 0;
		//cout << "i=" << i << endl;
		//cout << " =" << (int)number[i] << endl;
	}
}

void BigInteger::stuffVector(vector<char> vect) {
	for (int i = size - 1; i > 0; i--) {
		number[i] = vect.back();
		vect.pop_back();
	}
}

void BigInteger::shrink(int nuSize) {
	size = nuSize + 1;
	char* nuNumber = new char[size];
	for (int i = 0; i < size; i++) {
		nuNumber[i] = number[i];
	}
	delete number;
	number = nuNumber;
}

void BigInteger::resize() {
	int dec = 0;
	for (int i = 1; i < size; i++) {
		if (number[i] == 0) {
			dec++;
		}
		else {
			break;
		}
	}
	if (size - dec <= 1) {
		number[0] = '+';
		number[1] = 0;
		size = 2;
	}
	else if (dec != 0) {
		for (int i = 1; i < size - dec; i++) {
			number[i] = number[i + dec];
		}
		size -= dec;
	}
}

///
/// hipMemcpy wrapper methods
///

/**
 * Allocate and copy this BigInteger's number to device.
 */
char* BigInteger::copyNumberToDevice() const {
	char* d_number;
	hipMalloc( (void**) &d_number, sizeof(char) * size);
	hipMemcpy(d_number, number, sizeof(char) * size, hipMemcpyHostToDevice);
	return d_number;
}

void BigInteger::copyNumberFromDevice(char* d_number) {
	hipMemcpy(number, d_number, sizeof(char) * size, hipMemcpyDeviceToHost);
}

///
/// Carry the things
///

void BigInteger::applyAddCarry() {
	for (int i = size - 1; i >= 1; i--) {
		if (number[i] > 9) {
			number[i] -= 10;
			number[i - 1]++;
		}
	}
}

void BigInteger::applySubCarry() {
	for (int i = size - 1; i >= 0; i--) {
		if (number[i] < 0) {
			number[i] += 10;
			number[i - 1]--;
		}
	}
}

BigInteger BigInteger::applyMulCarry(int size_result, int size_first, int size_second) {
	int inc = 1;
	int i, n;
	char * tmp_result;
	BigInteger result(size_result);
	
	tmp_result = new char[size_result];

	for(i=1; i<size_result; i++)
		tmp_result[i] = 0;
	
	
	// i pour result, et n pour number
	for (i = size_result - 1, n = size_first*size_second-1; n > size_first; i--, n--) {
		if (n % size_first != 0) {
			tmp_result[i] += number[n];
			while (tmp_result[i]>9){
				tmp_result[i] -= 10;
				tmp_result[i-1]++;
			}
		}
		if(n % size_first == 0){
			i = size_result - inc;
			inc ++;
		}
	}
	// Signe du résultat
	tmp_result[0] = number[0];
	
	result.setNumber(tmp_result, size_result);

	delete [] tmp_result;
	return result;
}

///
/// Operations
///

BigInteger BigInteger::add(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();

	int size_b = size;
	if (other.size > size)
		size_b = other.size;
	
	BigInteger result(size_b + 1);
	char* d_newB = result.copyNumberToDevice();
	
	dim3 grid(1), block(size_b + 1);

	if (number[0] == other.number[0]) {	
		if (other.size > size) {
			kernel_add<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
		} else {
			kernel_add<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
		}
		result.copyNumberFromDevice(d_newB);
		result.applyAddCarry();
		if (number[0] == '-') {
			result.number[0] = '-';
		}
		else {
			result.number[0] = '+';
		}
	}
	else {
		// check where minus is
		if(other.number[0] == '-') {
			// check size of the number with minus
			if (isFirstBiggerThanSecond_2(number, other.number, size, other.size)) {
				kernel_sub<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
				result.copyNumberFromDevice(d_newB);
				result.applySubCarry();
			}
			else {
				kernel_sub<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
				result.copyNumberFromDevice(d_newB);
				result.applySubCarry();
				result.number[0] = '-';
			}

		}
		else {
			// check size of the number with minus
			if (isFirstBiggerThanSecond_2(number, other.number, size, other.size)) {
				kernel_sub<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
				result.copyNumberFromDevice(d_newB);
				result.applySubCarry();
				result.number[0] = '-';
			}
			else {
				kernel_sub<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
				result.copyNumberFromDevice(d_newB);
				result.applySubCarry();
			}
		}
	}

	result.resize();
	return result;
}

BigInteger BigInteger::substract(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();

	int size_b = size;
	if (other.size > size)
		size_b = other.size;
	
	BigInteger result(size_b + 1);
	char* d_newB = result.copyNumberToDevice();
	
	dim3 grid(1), block(size_b + 1);

	if (number[0] != other.number[0]) {
		if (isFirstBiggerThanSecond_2(number, other.number, size, other.size))
			kernel_add<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
		else
			kernel_add<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);

		result.copyNumberFromDevice(d_newB);
		result.applyAddCarry();
		if (number[0] == '-') {
			result.number[0] = '-';
		}
		else
				result.number[0] = '+';
	}
	else {
		if(isFirstBiggerThanSecond_2(number, other.number, size, other.size)) {
			kernel_sub<<<grid, block>>>(d_newB, d_number, d_other_number, size, size - other.size, &size_b);
			result.copyNumberFromDevice(d_newB);
			result.applySubCarry();
			if (number[0] == '-') {
				result.number[0] = '-';
			}
			else
				result.number[0] = '+';
		}
		else {
			kernel_sub<<<grid, block>>>(d_newB, d_other_number, d_number, other.size, other.size - size, &size_b);
			result.copyNumberFromDevice(d_newB);
			result.applySubCarry();
			if (number[0] == '+') {
				result.number[0] = '-';
			}
			else
				result.number[0] = '+';
		}
	}
	result.resize();
	return result;
}

BigInteger BigInteger::multiply(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();
	int size_newBbegin = size * other.size;
	
	BigInteger resultbegin(size_newBbegin);
	
	char* d_newB = resultbegin.copyNumberToDevice();
	
	dim3 grid(1), block(size, other.size);

	kernel_mul<<<grid, block>>>(d_newB, d_number, d_other_number, size, other.size, &size_newBbegin);

	resultbegin.copyNumberFromDevice(d_newB);

	hipFree(d_number);
	hipFree(d_other_number);
	hipFree(d_newB);

	int size_newB = size + other.size -1;
	BigInteger result(size_newB);
	result = resultbegin.applyMulCarry(size_newB, size, other.size);

	result.resize();
	return result;
}

BigInteger BigInteger::divide(const BigInteger& other) {
	if (other.size > size) {
		return BigInteger(2);
	}

	BigInteger result(size);
	if(number[0]=='-' || other.number[0]=='-')
		result.number[0]='-';
	else
		result.number[0]='+';

	int dividend_index = 1;
	vector<char> result_vector;
	BigInteger temp(other.size + 1);
	int rest_size = 0;
	int it_count = 0;
	do {
		it_count++;
		//cout << "___Iteration #" << it_count << endl;
		//cout << "beginning phase 1" << endl;
		int temp_index = rest_size + 1;
		for (int i = dividend_index; i < size; i++) {
			temp.number[temp_index] = number[i];
			temp_index++;
			dividend_index++;
			if (temp_index < other.size) {
				//still not big enough for the divisor
				continue;
			} else if (temp_index == other.size) {
				if (isFirstBiggerThanSecond(other.number, temp.number, other.size)) {
					// still too small, get one extra
					//TODO: handle potential "overflow"
					temp.number[temp_index] = number[i + 1];
					temp_index++;
					dividend_index++;
					break;
				} else {
					// temp is bigger than the divisor, go to next phase
					//cout << "shift right before"; temp.print();
					temp.shiftRight();
					//cout << "shift right after"; temp.print();
					break;
				}
			}
		}
		//cout << "end of phase 1, with temp: ";temp.print();
		//cout << "dividend_index is at " << dividend_index << "(=" << (int)number[dividend_index] << ")" << endl;
		//cout << "temp_index is at " << temp_index << endl;
		//cout << "beginning phase 2" << endl;
		BigInteger sub_res(other.size);
		char res = 0;
		do {
			sub_res = temp.substract(other);
			if (sub_res.number[0] == '+') {
				temp = sub_res;
				res++;
			}
		} while (sub_res.number[0] == '+');
		//cout << "end of phase 2, with res=" << (int)res << " and temp:"; temp.print();
		//cout << "beginning of phase 3" << endl;
		result_vector.push_back(res);
		temp.alignLeft(&rest_size);
		temp.shrink(rest_size + 1);
		//cout << "end of phase 3, with rest_size=" << rest_size << " and temp:"; temp.print();
		//cout << "loop condition: " << dividend_index << " < " << size  << " : " << (dividend_index <= size) << endl;
		//cout << "___end of iteration #" << it_count << endl;
	} while (dividend_index < size);

	// stuff vector into result
	//cout << "result_vector contents: ";
	for (int c = 0; c < result_vector.size(); c++) cout << (int) result_vector[c] << " ";
	cout << endl;
	result.stuffVector(result_vector);
	//cout << "Final result:"; result.print();
	return result;
}

BigInteger BigInteger::factorial() {
	BigInteger other;
	other.setNumber("+1", 2);
	other.zero();
	// MODIFY
//	int size_b = 100;
	BigInteger result;
	result.setNumber("+1", 2);
	result.zero();
	
	BigInteger tmp(2);
	tmp.setNumber("+1", 2);
	tmp.zero();
	char * d_tmp = tmp.copyNumberToDevice();
	while (isFirstBiggerThanSecond_2(number, other.number, size, other.size)) {
		other = other.add(tmp);
		result = result.multiply(other);
	}

	result.resize();
	return result;
}

BigInteger BigInteger::greatestCommonDivisor(const BigInteger& other) {
	char* d_number = copyNumberToDevice();
	char* d_other_number = other.copyNumberToDevice();

	BigInteger first_tmp(size);
	BigInteger second_tmp(other.size);
	bool nul, trouve;
	int i;

	trouve = false;

	int size_b = size;
	if (other.size > size)
		size_b = other.size;
	
	BigInteger tmp(size);
	BigInteger sub(size_b + 1);
	char* d_newB = sub.copyNumberToDevice();

	first_tmp.copyNumberFromDevice(d_number);
	second_tmp.copyNumberFromDevice(d_other_number);

	// Tant que le pgcd n'est pas trouvé on continue
	while(!trouve){
		nul = true;
		
		dim3 grid(1), block(size_b + 1);
		kernel_sub<<<grid, block>>>(d_newB, d_number, d_other_number, first_tmp.size, first_tmp.size - second_tmp.size, &size_b);

		sub.copyNumberFromDevice(d_newB);
		sub.applySubCarry();

		// Si la soustraction ne fait pas zéro on met nul à false
		for(i=1; i<size_b+1; i++){
			if(sub.number[i]!=0)
				nul = false;
		}

		// Si c'est nul on renvoie le pgcd
		if(nul || first_tmp.number[0]=='-'){
			trouve = true;
			//second_tmp.copyNumberFromDevice(d_other_number);
			return second_tmp;
		}
		// Si c'est positif
		else{
			// first = sub
			for(i=sub.size-1; i>0; i--){
				first_tmp.number[i] = sub.number[i];
			}
			first_tmp.number[0] = '+';
			first_tmp.size = sub.size;
			first_tmp.resize();
			second_tmp.resize();
			// Si second > first on les inverse
			if (!isFirstBiggerThanSecond_2(first_tmp.number, second_tmp.number, first_tmp.size, second_tmp.size)){
				for(i=first_tmp.size-1; i>-1; i--){
					tmp.number[i] = first_tmp.number[i];
				}
				tmp.size = first_tmp.size;
				for(i=second_tmp.size-1; i>-1; i--){
					first_tmp.number[i] = second_tmp.number[i];
				}
				first_tmp.size = second_tmp.size;
				for(i=tmp.size-1; i>-1; i--){
					second_tmp.number[i] = tmp.number[i];
				}
				second_tmp.size = tmp.size;
			}
			d_number = first_tmp.copyNumberToDevice();
			d_other_number = second_tmp.copyNumberToDevice();
			// On remet sub à 0
			for(i=0; i<sub.size; i++)
				sub.number[i] = 0;
			d_newB = sub.copyNumberToDevice();
			size_b = first_tmp.size;
			sub.size = size_b+1;
		}
	}
}




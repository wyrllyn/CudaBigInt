#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>
#include <cstdio>

#include "utility.h"

using namespace std;

__device__ inline void charAtomicAdd(char *address, char value) {
   int oldval, newval, readback;
 
   oldval = *address;
   newval = oldval + value;
   while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval) {
      oldval = readback;
      newval = oldval + value;
   }
}


///
/// Kernel functions
///

/*
How to use this method :
- newB and size newB contains the result
- size_biggest is the biggest size
- diff is the difference between sizes
- first is the biggest char*
- second is the other
- calcul newB avant

*/
__global__ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int i = threadIdx.x;
#if __CUDA_ARCH__>=200
	//printf("#threadIdx.x = %d\n", threadIdx.x);
#endif
	if (i == 0) return;

	//for (int i = size_biggest - 1; i >= 0; i--) {
	if (i - 1 - diff >= 0 && (second[i - 1 - diff] != '+' && second[i - 1 - diff] != '-')) {
		tmp = second[i - 1 - diff] + first[i - 1];
	} else if (first[i - 1] != '+' && first[i - 1] != '-') {
		tmp = first[i - 1];
	}

	if (tmp >= 10) {
		//charAtomicAdd(&newB[i], 1);
		newB[i - 1]++;
		tmp = tmp % 10;
	}
	if (i != 0)
		newB[i] += tmp;
	//}
}


__global__ void kernel_sub(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int i = threadIdx.x;
#if __CUDA_ARCH__>=200
	//printf("#threadIdx.x = %d\n", threadIdx.x);
#endif
	if (i == 0) return;

	//for (int i = size_biggest - 1; i >= 0; i--) {
	if (i - 1 - diff >= 0 && (second[i - 1 - diff] != '+' && second[i - 1 - diff] != '-')) {
		tmp = first[i - 1] - second[i-1-diff];
	} else if (first[i - 1] != '+' && first[i - 1] != '-') {
		tmp = first[i - 1];
	}

	if (tmp < 0) {
		// warning 10 - tmp ?
		newB[i - 1]--;
		tmp += 10;
	}
	if (i != 0)
		newB[i] += tmp;
	//}
}

// first is the bigInt with de biggest size
__global__ void kernel_mul(char* newB,  char* first, char* second, int size_first, int size_second, int * size_newB) {
/*#if __CUDA_ARCH__>=200
printf("une connerie%d\n", threadIdx.x);
#endif*/

	int index = 0;
	int tmp = 0;
	int tmp_second = 0;
	int carry = 0;
	int carry_second = 0;
	int i = threadIdx.x;
	int j = threadIdx.y;

	index = (j-1) + (i-1) + 2 ;

	//for (int i = size_second - 1; i >= 0; i--) {
	//index = (*size_newB) - size_second + i + (j - size_first);
#if __CUDA_ARCH__>=200
	//printf("#threadIdx.x (i) = %d\n", threadIdx.x);
	//printf("#threadIdx.y (j) = %d\n", threadIdx.y);
	//printf("#index = %d\n", index);
#endif
		//for (int j = size_first - 1; j >= 0 ; j--) {
	if(j!=0 && i!=0){
		tmp = first[i] * second[j] + carry;
		while (tmp >= 10) {
			tmp -= 10;
			carry++;
		}
		newB[index - 1] += carry;

		tmp_second = newB[index] + tmp;
		if (tmp_second >= 10) {
			tmp_second = tmp_second % 10;
			carry_second = 1;
		}
		newB[index - 1] += carry_second;
		newB[index] += tmp_second;
	}

	if(j==0 && i==0){
		if(first[j]=='-' || second[i]=='-')
			newB[0]='-';
		else
			newB[0]='+';
	}
		//}


		// add values : how ??
	//}
}



// first is the bigInt with de biggest size
/*__global__*/ /*void kernel_mul(char* newB,  char* first, char* second, int size_first, int size_second, int * size_newB) {
	(*size_newB) = size_first + size_second;
	init(*size_newB, newB);
	int index = 0;
	int tmp = 0;
	int tmp_second = 0;
	int carry = 0;
	int carry_second = 0;

	for (int i = size_second - 1; i >= 0; i--) {
		index = (*size_newB) - size_second + i ;
		for (int j = size_first - 1; j >= 0 ; j--) {
			//cout << "i = " << i << " j= " << j << " index = " << index << endl;
			tmp = first[j] * second[i] + carry;
			//cout << "1 tmp = " << tmp << endl;
			carry = 0;
			while (tmp >= 10) {
				tmp -= 10;
				carry++;
			}
		//	cout << "2 tmp = " << tmp << endl;
		//	cout << "carry = " << carry << endl;

			tmp_second = newB[index] + tmp + carry_second;
			if (tmp_second >= 10) {
		//		cout << " test second " << endl;
				tmp_second = tmp_second % 10;
				carry_second = 1;
			}
			newB[index] = tmp_second;
			index --;
			if (carry > 0 && j == 0) {
		//		cout << "test" << endl;
				newB[index] = carry;
			}
		}


		// add values : how ??
	}
}*/



/**
 * first is divided by second.
 * Note: since we are working with integers, the quotient can't be bigger than the dividend.
 * Also, if the divisor is bigger than the dividend, then the result is zero.
 */
void kernel_div(char* newB, const char* first, const char* second, int size_first, int size_second, int * size_newB) {
	if (size_first > size_second
		|| (size_first == size_second && isFirstBiggerThanSecond(first, second, size_first))
	) {
		*size_newB = size_first;
	} else {
		*size_newB = 1;
		init(*size_newB, newB);
		return;
	}


	char* temp = NULL;
	init(size_second + 1, temp);
	int t = 0; // temp's index
	int n = 0; // newB's index
	for (int i = size_first - 1; i >= 0; i -= t) {
		t = 0;
		for (int j = i - size_second; j <= i; j++) {
			if (j >= 0) {
				temp[t] = first[j];
				t++;
			}
		}
		// verify that we are not attempting to divide something too small
		if (isFirstBiggerThanSecond(second, temp, size_second)) {
			t = 0;
			for (int j = i - size_second - 1; j <= i; j++) {
				if (j < 0) {
					// nothing left to divide, exit function
					return;
				} else {
					temp[t] = first[j];
					t++;
				}
			}
		}
		// now that we have our thing, let's get to the division itself
		char res = 0;
		char* sub_res = NULL;
		int size_res = 0;
		init(size_second, sub_res);
		do {		
			//kernel_sub(sub_res, temp, second, size_second, size_second, &size_res);
			res++;
		} while (0); //sub_res > 0
		// current division done, save result & move on to the next
		newB[n] = res;
		n++;
	}
	// all divisions done, we need to realign our result;
	int diff = size_second - n;
	for (int i = size_second - 1; i > n; i++) {
		newB[i] = newB[i - diff];
	}
}





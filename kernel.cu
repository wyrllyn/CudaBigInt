#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>
#include <cstdio>

#include "utility.h"

using namespace std;

__device__ inline void charAtomicAdd(char *address, char value) {
   int oldval, newval, readback;
 
   oldval = *address;
   newval = oldval + value;
   while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval) {
      oldval = readback;
      newval = oldval + value;
   }
}


///
/// Kernel functions
///

/*
How to use this method :
- newB and size newB contains the result
- size_biggest is the biggest size
- diff is the difference between sizes
- first is the biggest char*
- second is the other
- calcul newB avant

*/
__global__ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int i = threadIdx.x;
#if __CUDA_ARCH__>=200
	//printf("#threadIdx.x = %d\n", threadIdx.x);
#endif
	if (i == 0) return;

	//for (int i = size_biggest - 1; i >= 0; i--) {
	if (i - 1 - diff >= 0 && (second[i - 1 - diff] != '+' && second[i - 1 - diff] != '-')) {
		tmp = second[i - 1 - diff] + first[i - 1];
	} else if (first[i - 1] != '+' && first[i - 1] != '-') {
		tmp = first[i - 1];
	}

	if (tmp >= 10) {
		//charAtomicAdd(&newB[i], 1);
		newB[i - 1]++;
		tmp = tmp % 10;
	}
	if (i != 0)
		newB[i] += tmp;
	//}
}

/// cpu function
/*__global__ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int carry = 0;
	(*size_newB) = size_biggest + 1;
	init(*size_newB, newB);
	int index = *size_newB - 1;
	int i = BlockIdx.x;

	for (int i = size_biggest - 1; i >= 0; i--) {
		if (i - diff >= 0 && (second[i] != '+' && second[i] != '-')) {
			tmp = second[i - diff] + first[i] + carry;
			cout << "__ " << (int) first[i] << " + " << (int) second[i - diff] << " + " << carry << " = " << tmp << endl;
		} else if (first[i] != '+' && first[i] != '-') {
			tmp = first[i] + carry;
			cout << "__ " << (int) first[i] << " + " << carry << " = " << tmp << endl;
		}

		if (tmp >= 10) {
			carry = 1;
			tmp = tmp % 10;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
		cout << index << "___ " << (int) newB[index] << endl;
		index--;
	}

	if (carry != 0) {
		newB[index] = carry;
		cout << index << "### " << (int) newB[index] << endl;
	}

	cout << "cheking final result" << endl;
	for (int i = 0; i < *size_newB; i++) {
		cout << (int) newB[i];
	}
	cout << endl;

}*/

/**
 * first is divided by second.
 * Note: since we are working with integers, the quotient can't be bigger than the dividend.
 * Also, if the divisor is bigger than the dividend, then the result is zero.
 */
void kernel_div(char* newB, const char* first, const char* second, int size_first, int size_second, int * size_newB) {
	if (size_first > size_second
		|| (size_first == size_second && isFirstBiggerThanSecond(first, second, size_first))
	) {
		*size_newB = size_first;
	} else {
		*size_newB = 1;
		init(*size_newB, newB);
		return;
	}


	char* temp = NULL;
	init(size_second + 1, temp);
	int t = 0; // temp's index
	int n = 0; // newB's index
	for (int i = size_first - 1; i >= 0; i -= t) {
		t = 0;
		for (int j = i - size_second; j <= i; j++) {
			if (j >= 0) {
				temp[t] = first[j];
				t++;
			}
		}
		// verify that we are not attempting to divide something too small
		if (isFirstBiggerThanSecond(second, temp, size_second)) {
			t = 0;
			for (int j = i - size_second - 1; j <= i; j++) {
				if (j < 0) {
					// nothing left to divide, exit function
					return;
				} else {
					temp[t] = first[j];
					t++;
				}
			}
		}
		// now that we have our thing, let's get to the division itself
		char res = 0;
		char* sub_res = NULL;
		int size_res = 0;
		init(size_second, sub_res);
		do {		
			//kernel_sub(sub_res, temp, second, size_second, size_second, &size_res);
			res++;
		} while (0); //sub_res > 0
		// current division done, save result & move on to the next
		newB[n] = res;
		n++;
	}
	// all divisions done, we need to realign our result;
	int diff = size_second - n;
	for (int i = size_second - 1; i > n; i++) {
		newB[i] = newB[i - diff];
	}
}

/*__global__*/ void kernel_sub(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {

	int tmp = 0;
	int carry = 0;
	(*size_newB) = size_biggest;
	init(*size_newB, newB);
	int index = *size_newB - 1;

	for (int i = size_biggest - 1; i >= 0; i--) {
		if (i - diff >= 0) {
			tmp = first[i] - second[i-diff] - carry;
			//cout << "__ " << (int) first[i] << " - " << (int) second[i - diff] << " - " << carry << " = " << tmp << endl;
		} else {
			tmp = first[i] - carry;
			//cout << "__ " << (int) first[i] << " - " << carry << " = " << tmp << endl;
		}

		if (tmp < 0) {
			// warning 10 - tmp ?
			carry = 1;
			tmp += 10 ;
		}
		else {
			carry = 0;
		}
		newB[index] = tmp;
		cout << "index : " << index << "___ " << (int) newB[index] << endl;
		index--;
	}

	*size_newB = update(newB, *size_newB);
	

	cout << "cheking final result" << endl;
	for (int i = 0; i < *size_newB; i++) {
		cout << (int) newB[i];
	}
	cout << endl;
}

// first is the bigInt with de biggest size
/*__global__*/ void kernel_mul(char* newB,  char* first, char* second, int size_first, int size_second, int * size_newB) {
	(*size_newB) = size_first + size_second;
	init(*size_newB, newB);
	int index = 0;
	int tmp = 0;
	int tmp_second = 0;
	int carry = 0;
	int carry_second = 0;

	for (int i = size_second - 1; i >= 0; i--) {
		index = (*size_newB) - size_second + i ;
		for (int j = size_first - 1; j >= 0 ; j--) {
			//cout << "i = " << i << " j= " << j << " index = " << index << endl;
			tmp = first[j] * second[i] + carry;
			//cout << "1 tmp = " << tmp << endl;
			carry = 0;
			while (tmp >= 10) {
				tmp -= 10;
				carry++;
			}
		//	cout << "2 tmp = " << tmp << endl;
		//	cout << "carry = " << carry << endl;

			tmp_second = newB[index] + tmp + carry_second;
			if (tmp_second >= 10) {
		//		cout << " test second " << endl;
				tmp_second = tmp_second % 10;
				carry_second = 1;
			}
			newB[index] = tmp_second;
			index --;
			if (carry > 0 && j == 0) {
		//		cout << "test" << endl;
				newB[index] = carry;
			}
		}


		// add values : how ??
	}
}



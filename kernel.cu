#include "hip/hip_runtime.h"
#include "kernel.h"

#include <iostream>
#include <cstdio>

#include "utility.h"

using namespace std;

__device__ inline void charAtomicAdd(char *address, char value) {
   int oldval, newval, readback;
 
   oldval = *address;
   newval = oldval + value;
   while ((readback=atomicCAS((int *)address, oldval, newval)) != oldval) {
      oldval = readback;
      newval = oldval + value;
   }
}


///
/// Kernel functions
///

/*
How to use this method :
- newB and size newB contains the result
- size_biggest is the biggest size
- diff is the difference between sizes
- first is the biggest char*
- second is the other
- calcul newB avant

*/
__global__ void kernel_add(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int i = threadIdx.x;
#if __CUDA_ARCH__>=200
	//printf("#threadIdx.x = %d\n", threadIdx.x);
#endif
	if (i == 0) return;

	//for (int i = size_biggest - 1; i >= 0; i--) {
	if (i - 1 - diff >= 0 && (second[i - 1 - diff] != '+' && second[i - 1 - diff] != '-')) {
		tmp = second[i - 1 - diff] + first[i - 1];
	} else if (first[i - 1] != '+' && first[i - 1] != '-') {
		tmp = first[i - 1];
	}

	if (tmp >= 10) {
		//charAtomicAdd(&newB[i], 1);
		newB[i - 1]++;
		tmp = tmp % 10;
	}
	if (i != 0)
		newB[i] += tmp;
	//}
}


__global__ void kernel_sub(char* newB, char* first, char* second, int size_biggest, int diff, int * size_newB) {
	int tmp = 0;
	int i = threadIdx.x;
#if __CUDA_ARCH__>=200
	//printf("#threadIdx.x = %d\n", threadIdx.x);
#endif
	if (i == 0) return;

	//for (int i = size_biggest - 1; i >= 0; i--) {
	if (i - 1 - diff >= 0 && (second[i - 1 - diff] != '+' && second[i - 1 - diff] != '-')) {
		tmp = first[i - 1] - second[i-1-diff];
	} else if (first[i - 1] != '+' && first[i - 1] != '-') {
		tmp = first[i - 1];
	}

	if (tmp < 0) {
		// warning 10 - tmp ?
		newB[i - 1]--;
		tmp += 10;
	}
	if (i != 0)
		newB[i] += tmp;
	//}
}

// first is the bigInt with the biggest size
__global__ void kernel_mul(char* newB,  char* first, char* second, int size_first, int size_second, int * size_newB) {

	int i = threadIdx.x;
	int j = threadIdx.y;

	int tid = j * gridDim.x * blockDim.x + i ;

	if(j!=0 && i!=0){
		newB[tid] = first[i] * second[j];
	}

	if(j==0 && i==0){
		if(first[j]=='-' || second[i]=='-')
			newB[0]='-';
		else
			newB[0]='+';
	}
}



/// Unused - see BigInteger::divide()
/**
 * first is divided by second.
 * Note: since we are working with integers, the quotient can't be bigger than the dividend.
 * Also, if the divisor is bigger than the dividend, then the result is zero.
 */
__global__ void kernel_div(char* newB, char* first, char* second, int size_first, int size_second, int * size_newB, char* aux) {
	int i = threadIdx.x;
	int j = threadIdx.y;

	if(j==0 && i==0){
		if(first[j]=='-' || second[i]=='-')
			newB[0]='-';
		else
			newB[0]='+';
		return;
	}

#if __CUDA_ARCH__>=200
	printf("#i, j = %d, %d\n", i, j);
#endif
	// adapted from kernel_sub
	int diff = size_first - size_second;
	int tmp = 0;
	if (j - 1 - diff >= 0 && (second[j - 1 - diff] != '+' && second[j - 1 - diff] != '-')) {
		tmp = first[j - 1] - second[j-1-diff];
	} else if (first[j - 1] != '+' && first[j - 1] != '-') {
		tmp = first[j - 1];
	}

	if (tmp < 0) {
		// warning 10 - tmp ?
		aux[i * size_first + j - 1]--;
		tmp += 10;
	}
	if (i != 0)
		aux[i * size_first + j] += tmp;
	// end of kernel_sub

#if __CUDA_ARCH__>=200
	printf("#aux = %d\n", aux[i * size_first + j]);
#endif

/*
	char* temp = NULL;
	//init(size_second + 1, temp);
	int t = 0; // temp's index
	int n = 0; // newB's index
	for (int i = size_first - 1; i >= 0; i -= t) {
		t = 0;
		for (int j = i - size_second; j <= i; j++) {
			if (j >= 0) {
				temp[t] = first[j];
				t++;
			}
		}
		// verify that we are not attempting to divide something too small
		if (isFirstBiggerThanSecond(second, temp, size_second)) {
			t = 0;
			for (int j = i - size_second - 1; j <= i; j++) {
				if (j < 0) {
					// nothing left to divide, exit function
					return;
				} else {
					temp[t] = first[j];
					t++;
				}
			}
		}
		// now that we have our thing, let's get to the division itself
		char res = 0;
		char* sub_res = NULL;
		int size_res = 0;
		//init(size_second, sub_res);
		do {		
			//kernel_sub(sub_res, temp, second, size_second, size_second, &size_res);
			res++;
		} while (0); //sub_res > 0
		// current division done, save result & move on to the next
		newB[n] = res;
		n++;
	}
	// all divisions done, we need to realign our result;
	int diff = size_second - n;
	for (int i = size_second - 1; i > n; i++) {
		newB[i] = newB[i - diff];
	}*/
}





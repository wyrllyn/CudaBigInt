#include "BigInteger.h"

#include <iostream>

using namespace std;


int main(int argc, char** argv) {
	cout << "This program assumes that its user enters coherent arguments (op number number), such as + -25 12" << endl;
	cout << "Enter boggus data at your own risks..." << endl;

	BigInteger left, right;
	OperationType opType;
	if (argc > 3)
		cout << argv[1] << " " << argv[2] << " " << argv[3] << endl;
	else
		cout << argv[1] << " " << argv[2] << endl;
	if (argc >= 2) {
		opType = identifyOperationType(argv[1]);
		left.setNumber(argv[2], string(argv[2]).size());
		left.zero();
		switch (opType) {
		case ADD:
		case SUBSTRACT:
		case MULTIPLY:
		case DIVIDE:
		case GCD:
			right.setNumber(argv[3], string(argv[3]).size());
			right.zero();
			break;
		case FACTORIAL:
			break;
		case ERROR:
			cout << "Unrecognised operation type: " << argv[1] << endl;
			break;
		}
	} else {
		cout << "Insufficient number of arguments" << endl;
	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	BigInteger result;
	int r;
	switch (opType) {
	case ADD:
		result = left.add(right);
		break;
	case SUBSTRACT:
		result = left.substract(right);
		break;
	case MULTIPLY:
		result = left.multiply(right);
		break;
	case DIVIDE:
		r = atoi(argv[3]);
		if (r == 0) {
			cout << "Nice try..." << endl;
			return 0;
		} else {
			result = left.divide(right);
		}
		break;
	case FACTORIAL:
		result = left.factorial();
		break;
	case GCD:
		result = left.greatestCommonDivisor(right);
		break;
	default:
		cout << "Reaching default case." << endl;
		break;
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	cout << "time: " << elapsed_time << " ms" << endl;
	result.print();
}



#include "BigInteger.h"

#include <iostream>

using namespace std;


int main(int argc, char** argv) {
	cout << "This program assumes that its user enters coherent arguments (op number number), such as * -25 12" << endl;
	cout << "Enter boggus data at your own risks..." << endl;

	BigInteger left, right;
	OperationType opType;
	cout << argv[1] << " " << argv[2] << " " << argv[3] << endl;
	if (argc >= 2) {
		opType = identifyOperationType(argv[1]);
		left.setNumber(argv[2], string(argv[2]).size());
		left.zero();
		switch (opType) {
		case ADD:
		case SUBSTRACT:
		case MULTIPLY:
		case DIVIDE:
			right.setNumber(argv[3], string(argv[3]).size());
			right.zero();
			break;
		case ERROR:
			cout << "Unrecognised operation type: " << argv[1] << endl;
			break;
		}
	} else {
		cout << "Insufficient number of arguments" << endl;
	}
	cout << "Here is what we have:" << endl;
	left.print();
	cout << argv[1] << endl;
	right.print();

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	switch (opType) {
	case ADD:
		left.add(right);
		break;
	case SUBSTRACT:
		left.substract(right);
		break;
	case MULTIPLY:
		left.multiply(right);
		break;
	case DIVIDE:
		left.divide(right);
		break;
	case FACTORIAL:
		left.factorial(right);
		break;
	case GCD:
		left.greatestCommonDivisor(right);
		break;
	default:
		cout << "Reaching default case." << endl;
		break;
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsed_time;
	hipEventElapsedTime(&elapsed_time, start, stop);
	cout << "time: " << elapsed_time << " ms" << endl;

	///
	/// Testing block
	///
	/*#define SIZE_FIRST 2
	#define SIZE_SECOND 2
	#define NU_SIZE 4
	char* nu = new char[NU_SIZE], * g_nu;
	char* first = new char[SIZE_FIRST];
	first[0] = 3; first[1] = 5;
	char* second = new char[SIZE_SECOND];
	second[0] = 2; second[1] = 0; // second[2] = 2;
	int nuSize = NU_SIZE;*/
	/*
	char* g_first, *g_second;
	hipMalloc( (void**) &g_first, sizeof(char) * 2 );
	hipMalloc( (void**) &g_second, sizeof(char) * 2 );
	hipMalloc( (void**) &g_nu, sizeof(char) * 2 );
	hipMemcpy(g_first, first, sizeof(char) * 2, hipMemcpyHostToDevice);
	hipMemcpy(g_second, second, sizeof(char) * 2, hipMemcpyHostToDevice);
	kernel_add<<<grid, block>>>(g_nu, g_first, g_second, 2, 2, &nuSize);
	hipMemcpy(nu, g_nu, sizeof(char) * 2, hipMemcpyDeviceToHost);
	*/
	//kernel_add(nu, first, second, SIZE_FIRST, SIZE_FIRST - SIZE_SECOND, &nuSize);
	//kernel_div(nu, first, second, SIZE_FIRST, SIZE_FIRST - SIZE_SECOND, &nuSize);
	//kernel_mul(nu, first, second, SIZE_FIRST, /*SIZE_FIRST -*/ SIZE_SECOND, &nuSize);
	/*for (int i = 0; i < SIZE_FIRST; i++) {
		cout << (int) first[i];
	}
	cout << " * ";
	for (int i = 0; i < SIZE_SECOND; i++) {
		cout << (int) second[i];
	}
	cout << " = ";
	for (int i = 0; i < nuSize; i++) {
		cout << (int) nu[i];
	}
	cout << endl;*/
	///
	/// End of testing block
	///
}


